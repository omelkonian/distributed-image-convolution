#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#define FRACTION_CEILING(numerator, denominator) ((numerator+denominator-1)/denominator)

/* Filter on GPU constant memory */
__constant__ int filterGPU[9];

/* Indexes a 2D array which is contiguously allocated in memory as a 1D array at position: (row, column) */
__device__ inline unsigned char* indexAt(unsigned char *array, int width, int bpp, int row, int col) {
	return &array[(row * width + col) * bpp];
}

__device__ inline int* indexAt2(int *array, int width, int bpp, int row, int col) {
	return &array[(row * width + col) * bpp];
}

/* Convolution on GPU (1 thread per pixel) */
__global__ void convolute(unsigned char *image, unsigned char *buffer, int sum, int height, int width, int bpp) { 	
	// Get thread's coordinates
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	int Y = threadIdx.y + blockIdx.y * blockDim.y;	

	// If it is inner pixel, convolute
	if ((X > 0) && (Y > 0) && (X < height - 1) && (Y < width - 1)) { 
		for (int offset = 0; offset < bpp; offset++) {			
			int newValue = 	(*(indexAt(image, width, bpp, X - 1	, Y - 1	) + offset)) * (*indexAt2(filterGPU, 3, 1, 0, 0))
						  +	(*(indexAt(image, width, bpp, X - 1	, Y    	) + offset)) * (*indexAt2(filterGPU, 3, 1, 0, 1))
						  + (*(indexAt(image, width, bpp, X - 1	, Y + 1	) + offset)) * (*indexAt2(filterGPU, 3, 1, 0, 2))
						  + (*(indexAt(image, width, bpp, X    	, Y - 1	) + offset)) * (*indexAt2(filterGPU, 3, 1, 1, 0))
						  + (*(indexAt(image, width, bpp, X  	, Y    	) + offset)) * (*indexAt2(filterGPU, 3, 1, 1, 1))
						  + (*(indexAt(image, width, bpp, X    	, Y + 1	) + offset)) * (*indexAt2(filterGPU, 3, 1, 1, 2))
						  + (*(indexAt(image, width, bpp, X + 1	, Y - 1	) + offset)) * (*indexAt2(filterGPU, 3, 1, 2, 0))
						  + (*(indexAt(image, width, bpp, X + 1	, Y    	) + offset)) * (*indexAt2(filterGPU, 3, 1, 2, 1))
						  + (*(indexAt(image, width, bpp, X + 1	, Y + 1	) + offset)) * (*indexAt2(filterGPU, 3, 1, 2, 2));

			newValue /= sum;

			if (newValue > UCHAR_MAX) newValue = UCHAR_MAX;
			else if (newValue < 0) newValue = 0;

			*indexAt(buffer, width, bpp, X, Y) = newValue;
		}
	}
	// Otherwise (if valid coordinates), just copy pixel
	else if ((X >= 0) && (X < height) && (Y >= 0) && (Y < width))
		*indexAt(buffer, width, bpp, X, Y) = *indexAt(image, width, bpp, X, Y);	
}

extern "C" void initiate(int height, int width, int blockSize, int matrixSize, unsigned char *input, unsigned char *output, int loops, int sum, int bpp, int *filterCPU){
	/* Declarations */
	unsigned char *image, *buffer;
	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));

	/* Allocate GPU matrices (image, buffer) */
	CUDA_SAFE_CALL(hipMalloc((void**) &image, matrixSize));
	CUDA_SAFE_CALL(hipMalloc((void**) &buffer, matrixSize));

	/* Calculate grid size */
	int gridX = FRACTION_CEILING(height, blockSize);
	int gridY = FRACTION_CEILING(width, blockSize);

	dim3 block(blockSize, blockSize);
	dim3 grid(gridX, gridY);

	/* Copy filter from CPU to GPU (constant memory) */
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(filterGPU), filterCPU, 9 * sizeof(int)));

	/* Initialize image */
	CUDA_SAFE_CALL(hipMemcpy(image, input, matrixSize, hipMemcpyHostToDevice));	


	/* Start time */
	CUDA_SAFE_CALL(hipEventRecord(start, 0));

	/* Main Loop (No convergence check) */
	for (int loop = 0; loop < loops; loop++) {
		// Convolution on GPU
		convolute<<<grid, block>>>(image, buffer, sum, height, width, bpp);
		CUDA_SAFE_CALL(hipGetLastError());
		
		// Synchronize threads
		CUDA_SAFE_CALL(hipDeviceSynchronize());

		// Swap buffers
		unsigned char *temp = image;
		image = buffer;
		buffer = temp;
	}

	/* Stop time */
	CUDA_SAFE_CALL(hipEventRecord(stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(stop));

	float elapsedTime;
	CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("%3.1f ms\n", elapsedTime);
	

	/* Copy GPU's image to CPU's output */
	CUDA_SAFE_CALL(hipMemcpy(output, image, matrixSize, hipMemcpyDeviceToHost));

	/* Free resources */
	CUDA_SAFE_CALL(hipFree(image));
	CUDA_SAFE_CALL(hipFree(buffer));

	CUDA_SAFE_CALL(hipEventDestroy(start));
	CUDA_SAFE_CALL(hipEventDestroy(stop));
}
